
#include <hip/hip_runtime.h>
extern "C"   
{
  __global__ void DYbinaryentropy_32(const int lengthX, const float *x,  const float *y, const float *t, float *z)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<lengthX)
      {
	z[i] += t[0]*((y[i]-x[i])/(y[i]*(1.0-y[i])))/lengthX;
      }
    }
}