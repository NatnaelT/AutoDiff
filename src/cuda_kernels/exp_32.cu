
#include <hip/hip_runtime.h>
extern "C"  
{
    __global__ void expkernel_32(const int lengthA, const float *a,  float *b)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthA)
	{
	  b[i] = exp(a[i]); 
	}
    }
}