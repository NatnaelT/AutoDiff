
#include <hip/hip_runtime.h>
extern "C"  
{
    __global__ void expkernel(const int lengthA, const double *a,  double *b)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthA)
	{
	  b[i] = exp(a[i]); 
	}
    }
}