
#include <hip/hip_runtime.h>
extern "C"   
{
  __global__ void DmeanSquareLoss_32(const int lengthx, const float pref, const float *gradc, const float *x,const float *y, float *gradn )
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthx)
	  {
	    gradn[i] += pref * gradc[0] * (x[i]-y[i]);
	  }	
    }
}