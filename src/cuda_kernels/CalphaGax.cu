
#include <hip/hip_runtime.h>
extern "C"  
{
    __global__ void CalpahGax(const double alpha, const double *a, const double *b, double *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        c[i] = alpha*a[0]*b[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
    }
}