
#include <hip/hip_runtime.h>
extern "C"  
{
  __global__ void gradalex(const int n, const double *a, const double *b, double *c)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<n)
      {
	if (b[i]>-0.5)
	  {c[i] += a[i];}	
	else	 
	  {c[i] -= 0.5*a[i]/b[i];}
      }	
  }
}