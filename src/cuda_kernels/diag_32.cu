
#include <hip/hip_runtime.h>
// B=diag(A)

extern "C"   
{
  __global__ void diag_kernel_32(const int lengthA, const float *a, float *b)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthA)
	  {
	    b[i]=a[i+i*lengthA];
	  }
    }
}