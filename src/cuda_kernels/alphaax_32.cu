
#include <hip/hip_runtime.h>
extern "C"  
{
    __global__ void alphaax_32(const int lengthC, const float alpha, const float *a, const float *b, float *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        if (i<lengthC)
        {	
        c[i] = alpha*a[0]*b[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
        }
    }
}