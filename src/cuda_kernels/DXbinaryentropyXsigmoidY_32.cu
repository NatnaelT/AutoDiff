
#include <hip/hip_runtime.h>
extern "C"   
{
  __global__ void DXbinaryentropyXsigmoidY_32(const int lengthX, const float *x,  const float *y, const float *t, float *z)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<lengthX)
      {
	z[i] -= t[0]*(y[i]-log(x[i]/(1.0-x[i])))/lengthX;
      }
    }
}