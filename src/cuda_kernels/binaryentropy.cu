
#include <hip/hip_runtime.h>
extern "C"   
{
  __global__ void binaryentropy(const int lengthX, const double *x,  const double *y, double *z)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<lengthX)
      {
	z[i] = x[i]*log(x[i]/y[i])+ (1.0-x[i])*log((1.0-x[i])/(1.0-y[i]));
	}
    }
}