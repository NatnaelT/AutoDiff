
#include <hip/hip_runtime.h>
extern "C"  
{
  __global__ void gradalex_32(const int n, const float *a, const float *b, float *c)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<n)
      {
	if (b[i]>-0.5)
	  {c[i] += a[i];}	
	else	 
	  {c[i] -= 0.5*a[i]/b[i];}
      }	
  }
}