
#include <hip/hip_runtime.h>
extern "C"  
{
  __global__ void alex(const int n, const double *a, double *b)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<n)
      {
	if (a[i]>-0.5)
	  {b[i] = a[i];}	
	else	 
	  {b[i] = -0.5*log(-a[i])-0.5*(1-log(0.5));}
      }	
  }
}