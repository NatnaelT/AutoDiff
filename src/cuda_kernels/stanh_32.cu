
#include <hip/hip_runtime.h>
extern "C"   
{
  __global__ void stanh_32(const int lengthA, const float alpha, const float *a, float *b)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthA)
	{
	  b[i] = alpha*tanh(a[i]);
	}
    }
}