
#include <hip/hip_runtime.h>
// filename: vsquare.cu
// a simple CUDA kernel to element multiply vector with itself

extern "C"   // ensure function name to be exactly "vsquare"
{
    __global__ void vsquare(const double *a, double *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        c[i] = a[i] * a[i];
    }
}