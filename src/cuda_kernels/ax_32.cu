
#include <hip/hip_runtime.h>
// filename: ax.cu
// a simple CUDA kernel to add two vectors

extern "C"   
{
    __global__ void ax_32(const int lengthC, const float a, const float *b, float *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthC)
	{
        c[i] = a*b[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
	}
    }
}