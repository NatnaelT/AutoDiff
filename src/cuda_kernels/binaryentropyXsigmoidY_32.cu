
#include <hip/hip_runtime.h>
extern "C"   
{
  __global__ void binaryentropyXsigmoidY_32(const int lengthX, const float *x,  const float *y, float *z)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<lengthX)
      {
	z[i]=x[i]*log(x[i])+(1.0-x[i])*log(1.0-x[i])-x[i]*y[i]+log(1.0+exp(y[i]));
      }
  }
}