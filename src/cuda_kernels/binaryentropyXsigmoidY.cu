
#include <hip/hip_runtime.h>
extern "C"   
{
  __global__ void binaryentropyXsigmoidY(const int lengthX, const double *x,  const double *y, double *z)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<lengthX)
      {
	z[i]=x[i]*log(x[i])+(1.0-x[i])*log(1.0-x[i])-x[i]*y[i]+log(1.0+exp(y[i]));
      }
  }
}