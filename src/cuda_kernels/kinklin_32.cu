
#include <hip/hip_runtime.h>
extern "C"  
{
  __global__ void kinklin_32(const int n, const float gamma, const float *a, float *b)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<n)
      {
	if (a[i]>0.0)
	  {b[i] = a[i];}	
	else	 
	  {b[i] = gamma*a[i];}
      }	
  }
}