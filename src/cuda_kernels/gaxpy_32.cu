
#include <hip/hip_runtime.h>
// filename: gaxpy.cu
// a simple CUDA kernel to add two vectors

extern "C"  
{
    __global__ void gaxpy_32(const int lengthC, const float *a, const float *b, float *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthC)
	{
        c[i] = a[0]*b[i] + c[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!	
	}
    }
}