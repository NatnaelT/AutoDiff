
#include <hip/hip_runtime.h>
extern "C"  
{
    __global__ void vabs(const int n, const double *a, double *b)
    {	       
      int i =	 threadIdx.x + blockIdx.x * blockDim.x;
      if (i<n) 
	{b[i]=fabs(a[i]);}
    }	  
}
