
#include <hip/hip_runtime.h>
// filename: gaxpy.cu
// a simple CUDA kernel to add two vectors

extern "C"   // ensure function name to be exactly "gaxpy"
{
    __global__ void gaxpy(const int lengthC, const double *a, const double *b, double *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthC)
	{
        c[i] = a[0]*b[i] + c[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!	
	}
    }
}