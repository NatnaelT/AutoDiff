
#include <hip/hip_runtime.h>
// B=diagm(A)

extern "C"   
{
  __global__ void diagm_kernel_32(const int lengthA, const float *a, float *b)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthA)
	  {
	    b[i+i*lengthA] = a[i];
	  }
    }
}