
#include <hip/hip_runtime.h>
extern "C"  
{
  __global__ void rectlin_32(const int n, const float *a, float *b)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<n)
      {
	if (a[i]>0.0)
	  {b[i] = a[i];}	
	else	 
	  {b[i] = 0.0;}
      }	
  }
}