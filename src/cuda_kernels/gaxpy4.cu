
#include <hip/hip_runtime.h>
// filename: gaxpy2.cu
// a simple CUDA kernel to add two vectors

extern "C"   // ensure function name to be exactly "gaxpy2"
{
    __global__ void gaxpy4(const int n, const double *a, const double *b, double *c)
    {
        int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < n) {
 c[i] = (double) i;  // REMEMBER ZERO INDEXING IN C LANGUAGE!!			
}		   

    }
}