
#include <hip/hip_runtime.h>
extern "C"  
{
    __global__ void CalpahGaxpGy(const double alpha, const double *a, const double *b, double *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        c[i] = alpha*a[0]*b[i]+c[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
    }
}