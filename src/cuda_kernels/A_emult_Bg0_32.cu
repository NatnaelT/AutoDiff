
#include <hip/hip_runtime.h>
extern "C"  
{
  __global__ void A_emult_Bg0_32(const int n, const float *a, const float *b, float *c)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<n)
      {
	if (b[i]>0.0)
	  {c[i] += a[i];}	
	else	 
	  {c[i] += 0.0;}
      }	
  }
}