
#include <hip/hip_runtime.h>
extern "C"   
{
  __global__ void Dstanh(const int lengthX, const double sf, const double *gradc, const double *fc,  double *gradn)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<lengthX)
      {
	gradn[i] += sf*gradc[i]*(1.0-(fc[i]/sf)*(fc[i]/sf));
      }
    }
}