
#include <hip/hip_runtime.h>
extern "C"  
{
  __global__ void xsigny_update_32(const int n, const float *a, float *b, float *c)
  {	       
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<n) 
      {
	if (b[i]>0) 
	  {c[i]+=a[i];}
	else
	  {if (b[i]<0) 
	      {c[i]-=a[i];}
	  }
      }
  }
}
