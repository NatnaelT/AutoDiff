
#include <hip/hip_runtime.h>
// filename: gaxpy2.cu
// a simple CUDA kernel to add two vectors

extern "C"   // ensure function name to be exactly "gaxpy2"
{
    __global__ void gaxpy3(const int n, const double *a, const double *b, double *c)
    {
        int i = threadIdx.x + threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
	if (i < n) {
printf("%d\n",i);
 c[i] = a[0]*b[i] + c[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!			
}		   

    }
}