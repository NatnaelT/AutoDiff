
#include <hip/hip_runtime.h>
// filename: gax.cu
// a simple CUDA kernel to add two vectors

extern "C"   // ensure function name to be exactly "gax"
{
    __global__ void gax(const int lengthC, const double *a, const double *b, double *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthC)
	{
        c[i] = a[0]*b[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
	}
    }
}