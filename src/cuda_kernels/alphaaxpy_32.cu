
#include <hip/hip_runtime.h>
extern "C"  
{
    __global__ void alphaaxpy_32(const int lengthC, const float alpha, const float *a, const float *b, float *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        if (i<lengthC)
        {	
        c[i] = alpha*a[0]*b[i]+c[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
        }
    }
}