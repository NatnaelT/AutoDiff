
#include <hip/hip_runtime.h>
// filename: vsquare.cu
// a simple CUDA kernel to element multiply vector with itself

extern "C"   // ensure function name to be exactly "vsquare"
{
    __global__ void vsquare(const double *a, double *c)
    {
        int i = threadIdx.x+blockIdx.x*blockDim.x;
	double v = a[i];
        c[i] = v*v;
   }
}