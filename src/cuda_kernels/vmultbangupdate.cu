
#include <hip/hip_runtime.h>
// filename: vmult!.cu
// a simple CUDA kernel to element multiply two vectors C=alpha*A.*B

extern "C"   // ensure function name to be exactly "vmultbang"
{
  __global__ void vmultbangupdate(const int lengthA, const double alpha, const double *a, const double *b, double *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthA)
	  {
	    c[i] += alpha*a[i] * b[i];
	  }	
    }
}