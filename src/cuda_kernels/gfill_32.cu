
#include <hip/hip_runtime.h>
extern "C"  
{
    __global__ void gfill_32(const int n, const float *a, float *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<n)
	{
        c[i] = a[0];
	}
    }
}