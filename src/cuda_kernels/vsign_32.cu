
#include <hip/hip_runtime.h>
extern "C"  
{
    __global__ void vsign_32(const int n, const float *a, float *b)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<n)
	  {
	    if (a[i]<0)
	      {b[i]=-1.0;}
	    else
	      {if (a[i]>0)
		  {b[i]=1.0;}
		else
		  {b[i]=0.0;}		    
	      }
	  }
    }
}