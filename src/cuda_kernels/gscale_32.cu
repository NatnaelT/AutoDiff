
#include <hip/hip_runtime.h>
extern "C"  
{
    __global__ void gscale_32(const int lengthB, const float *a, float *b)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthB)
	{
        b[i] = a[0]*b[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
	}
    }
}