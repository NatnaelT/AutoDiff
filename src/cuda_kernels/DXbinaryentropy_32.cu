
#include <hip/hip_runtime.h>
extern "C"   
{
  __global__ void DXbinaryentropy_32(const int lengthX, const float *x,  const float *y, const float *t, float *z)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<lengthX)
      {
	z[i] += t[0]*log(x[i]*(1.0-y[i])/(y[i]*(1.0-x[i])))/lengthX;
      }
    }
}