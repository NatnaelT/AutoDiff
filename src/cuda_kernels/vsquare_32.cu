
#include <hip/hip_runtime.h>
extern "C"   
{
    __global__ void vsquare_32(const float *a, float *c)
    {
        int i = threadIdx.x+blockIdx.x*blockDim.x;
	double v = a[i];
        c[i] = v*v;
   }
}