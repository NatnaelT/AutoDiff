
#include <hip/hip_runtime.h>
extern "C"   
{
  __global__ void tx1mx_32(const int lengthX, const float *t, const float *x,  float *z)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<lengthX)
      {
	z[i] += t[i]*x[i]*(1.0-x[i]);
      }
    }
}