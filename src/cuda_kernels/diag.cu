
#include <hip/hip_runtime.h>
// B=diag(A)

extern "C"   
{
  __global__ void diag_kernel(const int lengthA, const double *a, double *b)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthA)
	  {
	    b[i]=a[i+i*lengthA];
	  }
    }
}