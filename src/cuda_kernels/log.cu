
#include <hip/hip_runtime.h>
extern "C"  
{
    __global__ void logkernel(const int lengthA, const double *a,  double *b)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthA)
	{
	  b[i] = log(a[i]); 
	}
    }
}