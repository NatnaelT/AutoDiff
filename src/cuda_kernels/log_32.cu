
#include <hip/hip_runtime.h>
extern "C"  
{
    __global__ void logkernel_32(const int lengthA, const float *a,  float *b)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthA)
	{
	  b[i] = log(a[i]); 
	}
    }
}