
#include <hip/hip_runtime.h>
extern "C"   
{
    __global__ void sigmoid32(const int lengthA, const float *a,  float *b)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthA)
	{
	  b[i] = exp(-a[i]);  
	  b[i] = 1.0/(1.0+b[i]);
	}
    }
}