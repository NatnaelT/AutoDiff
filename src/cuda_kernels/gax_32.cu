
#include <hip/hip_runtime.h>
// filename: gax.cu
// a simple CUDA kernel to add two vectors

extern "C"  
{
    __global__ void gax_32(const int lengthC, const float *a, const float *b, float *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthC)
	{
        c[i] = a[0]*b[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
	}
    }
}