
#include <hip/hip_runtime.h>
extern "C"   
{
  __global__ void binaryentropy_32(const int lengthX, const float *x,  const float *y, float *z)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<lengthX)
      {
	z[i] = x[i]*log(x[i]/y[i])+ (1.0-x[i])*log((1.0-x[i])/(1.0-y[i]));
	}
    }
}