
#include <hip/hip_runtime.h>
// filename: gaxpy2.cu
// a simple CUDA kernel to add two vectors

extern "C"   // ensure function name to be exactly "gaxpy2"
{
    __global__ void gaxpy2(const double *a, const double *b, double *c)
    {
        int i = threadIdx.x + threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
        c[i] = a[0]*b[i] + c[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
    }
}