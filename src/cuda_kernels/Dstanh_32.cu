
#include <hip/hip_runtime.h>
extern "C"   
{
  __global__ void Dstanh_32(const int lengthX, const float sf, const float *gradc, const float *fc,  float *gradn)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<lengthX)
      {
	gradn[i] += sf*gradc[i]*(1.0-(fc[i]/sf)*(fc[i]/sf));
      }
    }
}