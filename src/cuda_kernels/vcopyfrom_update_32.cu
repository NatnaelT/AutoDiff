
#include <hip/hip_runtime.h>
extern "C"  
{
  __global__ void vcopyfrom_update_32(const int n, const int shift, const float *a, float *b)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<n)
      {
	b[i] += a[i+shift];
      }
  }
}