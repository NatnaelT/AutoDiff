
#include <hip/hip_runtime.h>
extern "C"   
{
  __global__ void DYbinaryentropyXsigmoidY_32(const int lengthX, const float *x, const float *y, const float *t, float *z)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<lengthX)
      {
	z[i] += t[0]*(1.0/(1.0+exp(-y[i]))-x[i])/lengthX;
      }
    }
}